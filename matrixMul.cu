#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication which makes use of shared memory
 * to ensure data reuse, the matrix multiplication is done using tiling approach.
 * It has been written for clarity of exposition to illustrate various CUDA programming
 * principles, not with the goal of providing the most performant generic kernel for matrix multiplication.
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> __global__ void MatrixMulCUDA(float *C, float *A,
                                                        float *B, int wA,
                                                        int wB) {
    // Block index
    //块坐标
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    //线程坐标
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;//A中该行起始对应的一维数组位置

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;//A中该行终点对应一维数组位置

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;//A子矩阵基本计算单位

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;//B中该列起始对应一维数组位置

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;//B子矩阵基本计算单位

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    //是计算的一行*一列
    for (int a = aBegin, b = bBegin;
            a <= aEnd;
            a += aStep, b += bStep) {
        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];//计算具体元素位置，1个thread表示1个元素
        Bs[ty][tx] = B[b + wB * ty + tx];//同上

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

	//以块为单位计算,直至A中该行计算完毕(对应大循环)
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;//对应元素位置
}

void ConstantInit(float *data, int size, float val) {
    for (int i = 0; i < size; ++i) {
        data[i] = val;
    }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char **argv,
                   int block_size, const dim3 &dimsA,
                   const dim3 &dimsB) {
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = reinterpret_cast<float *>(malloc(mem_size_A));//矩阵A内存首地址
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = reinterpret_cast<float *>(malloc(mem_size_B));//矩阵B内存首地址

    // Initialize host memory
    const float valB = 0.01f;
    ConstantInit(h_A, size_A, 1.0f);
    ConstantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);//结果矩阵
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = reinterpret_cast<float *>(malloc(mem_size_C));//矩阵C内存首地址

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

    // Setup execution parameters
    dim3 threads(block_size, block_size);//1个block里面(32*32)的线程
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);//1个grid里面这么多的block

    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    //预热，初步认为是矩阵乘法速度过快，不好计算时间,因此下面有300次循环
    if (block_size == 16) {
        MatrixMulCUDA<16> <<< grid, threads >>>(d_C, d_A, d_B,
                                                dimsA.x, dimsB.x);
    } else {
        MatrixMulCUDA<32> <<< grid, threads >>>(d_C, d_A, d_B,
                                                dimsA.x, dimsB.x);
    }

    printf("done\n");

    hipDeviceSynchronize();

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    checkCudaErrors(hipEventCreate(&start));

    hipEvent_t stop;
    checkCudaErrors(hipEventCreate(&stop));

    // Record the start event
    checkCudaErrors(hipEventRecord(start, NULL));

    // Execute the kernel
    int nIter = 300;

    for (int j = 0; j < nIter; j++) {
        if (block_size == 16) {
            MatrixMulCUDA<16> <<< grid, threads >>>(d_C, d_A, d_B,
                                                    dimsA.x, dimsB.x);
        } else {
            MatrixMulCUDA<32> <<< grid, threads >>>(d_C, d_A, d_B,
                                                    dimsA.x, dimsB.x);
        }
    }

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, NULL));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
                               static_cast<double>(dimsA.y) *
                               static_cast<double>(dimsB.x);
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) /
                       (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops," \
        " WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);

    // Copy result from device to host
    checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6;  // machine zero

    for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;

        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                   i, h_C[i], dimsA.x * valB, eps);
            correct = false;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    printf("\nNOTE: The CUDA Samples are not meant for performance"\
           "measurements. Results may vary when GPU Boost is enabled.\n");

    if (correct) {
        return EXIT_SUCCESS;
    } else {
        return EXIT_FAILURE;
    }
}


/**
 * Program main
 */
int main(int argc, char **argv) {
    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
            checkCmdLineFlag(argc, (const char **)argv, "?")) {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("  Note: Outer matrix dimensions of A & B matrices" \
               " must be equal.\n");

        exit(EXIT_SUCCESS);
    }

    // This will pick the best possible CUDA capable device, otherwise
    // override the device ID based on input provided at the command line
    int dev = findCudaDevice(argc, (const char **)argv);

    int block_size = 32;

    dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
    dim3 dimsB(5 * 4 * block_size, 5 * 2 * block_size, 1);

    // width of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "wA")) {
        dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
    }

    // height of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "hA")) {
        dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
    }

    // width of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "wB")) {
        dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
    }

    // height of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "hB")) {
        dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
    }

    if (dimsA.x != dimsB.y) {
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
               dimsA.x, dimsB.y);
        exit(EXIT_FAILURE);
    }

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
                                               dimsB.x, dimsB.y);

    int matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB);//dimsA.x表示wA，dimsA.y表示hA以此类推,边长最好是32的整数倍，否则1个线程束中会有空闲线程

    exit(matrix_result);
}

