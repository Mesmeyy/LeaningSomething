#include <stdio.h>
#include <errno.h>
#include<time.h>
#include<hip/hip_runtime.h>



#define BLOCK_NUM 32   //������
#define THREAD_NUM 256 // ÿ�����е��߳���

#define R_SIZE BLOCK_NUM * THREAD_NUM
//#define R_SIZE 4096
//#define R_SIZE 10

#define M_SIZE R_SIZE * R_SIZE

__global__ void mat_mul(int *mat1, int *mat2, int *result) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    // ÿ���̼߳���һ��
    const int row = bid * THREAD_NUM + tid;
    for (int c = 0; c < R_SIZE; c++) {
        for (int n = 0; n < R_SIZE; n++) {
            result[row*R_SIZE+c] += mat1[row*R_SIZE+n] * mat2[n*R_SIZE+c];
        }
    }
}

int main(int argc, char *argv[]) {
    int num;
    hipDeviceProp_t prop;
    hipError_t cudaStatus;
    cudaStatus = hipGetDeviceCount(&num);
    printf("deviceCount := %d\n",num);
    for(int i=0;i<num;i++){
            printf("i = %d\n",i);
            hipGetDeviceProperties(&prop,i);
            printf("name:%s\n",prop.name);
            printf("totalGlobalMem:%d\n",prop.totalGlobalMem);
            printf("totalGlobalMem:%d\n",prop.totalGlobalMem/1024);
            printf("totalGlobalMem:%d\n",prop.totalGlobalMem/1024/1024);
            printf("totalGlobalMem:%d\n",prop.totalGlobalMem/1024/1024/1024);
            printf("multiProcessorCount:%d\n",prop.multiProcessorCount);
            printf("maxThreadsPerBlock:%d\n",prop.maxThreadsPerBlock);
            printf("major:%d,minor:%d\n",prop.major,prop.minor);
            printf("  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n",
		          prop.pciDomainID, prop.pciBusID, prop.pciDeviceID);

    }
    cudaStatus=hipSetDevice(0);
    printf("return : %d\n",cudaStatus);
    cudaStatus = hipSetDevice(0);
    printf("return : %d\n",cudaStatus);
    //cudaStatus=cudaSetDevice(1);
    //printf("return : %d\n",cudaStatus);
    int *mat1, *mat2, *result;
    int *g_mat1, *g_mat2, *g_mat_result;
    FILE *fp;
    char path[256]="\0";
    char buf[R_SIZE*16];
    int i=0;
    clock_t start,finish; // �������
    double time;
    
    start=clock();     //start file copy
    
    if(argc!=2){
    	strcpy(path,"/data/006zzy/files/big_MM/bid_MM.txt");
    }else
        strcpy(path,argv[1]);
        
    fp = fopen(path, "r");
    if (fp == NULL)
    {
        printf("file is error=%d.",errno);
        return -1;
    }
    
    // ��һλ�����ʾ��ά����
    mat1 = (int*) malloc(M_SIZE * sizeof(int));
    mat2 = (int*) malloc(M_SIZE * sizeof(int));
    result = (int*) malloc(M_SIZE * sizeof(int));

    // initialize
    /*
    for (i = 0; i < M_SIZE; i++) {
        mat1[i] = rand()/1000000;
        mat2[i] = rand()/1000000;
        result[i] = 0;
        
    }*/
    i=0;
    //int j=0;
    while(fgets(buf,sizeof(buf),fp)){
       //printf(" --%d ---%s\n",j++,buf);
       char *s =NULL;
       s=strtok(buf," \t\n");
       while(s != NULL)
       {
           //if(i>M_SIZE) break;
           mat1[i++] =atoi(s);
           s = strtok(NULL," \t\n");
       }
       buf[0]='\0';
    }
    //fclose(fp);
    
    //printf(" %d \n",mat1[M_SIZE-1]);
    i=0;
    //int j=0;
    while(fgets(buf,sizeof(buf),fp)){
       //printf(" --%d ---%s\n",j++,buf);
       char *s =NULL;
       s=strtok(buf," \t\n");
       while(s != NULL)
       {
           //if(i>M_SIZE) break;
           mat2[i++] =atoi(s);
           s = strtok(NULL," \t\n");
       }
       buf[0]='\0';
    }
    fclose(fp);
    finish=clock();    //file read over
    time=(double)(finish-start)/CLOCKS_PER_SEC;//time for file read
    
    start=clock();     //start count    

    hipMalloc((void **)&g_mat1, sizeof(int) * M_SIZE);
    hipMalloc((void **)&g_mat2, sizeof(int) * M_SIZE);
    hipMalloc((void **)&g_mat_result, sizeof(int) * M_SIZE);

    hipMemcpy(g_mat1, mat1, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(g_mat2, mat2, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);

    mat_mul<<<BLOCK_NUM, THREAD_NUM>>>(g_mat1, g_mat2, g_mat_result);

    hipMemcpy(result, g_mat_result, sizeof(int) * M_SIZE, hipMemcpyDeviceToHost);
    hipFree(g_mat1);
    hipFree(g_mat2);
    hipFree(g_mat_result);
    free(mat1);
    free(mat2);
    free(result);
    finish=clock();    //����
    time=(double)(finish-start)/CLOCKS_PER_SEC;//��������ʱ��
    printf("time=%lf\n",time);//�������ʱ��
    return 0;
}
